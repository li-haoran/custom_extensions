#include "hip/hip_runtime.h"
/**********************************
 * Original Author: Haoqiang Fan
 * Modified by: Kaichun Mo
 *********************************/

#ifndef _EMD_KERNEL
#define _EMD_KERNEL

#include <cmath>
#include <vector>

#include <ATen/ATen.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>  // at::cuda::getApplyGrid
#include <THC/THC.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.type().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)


/********************************
* Forward kernel for approxmatch
*********************************/

template<typename scalar_t>
__global__ void approxmatch(int b,int n,int m,const scalar_t * __restrict__ p1,const scalar_t * __restrict__ p2,scalar_t * __restrict__ match,scalar_t * temp){
	scalar_t * remainL=temp+blockIdx.x*(n+m)*2, * remainR=temp+blockIdx.x*(n+m)*2+n,*ratioL=temp+blockIdx.x*(n+m)*2+n+m,*ratioR=temp+blockIdx.x*(n+m)*2+n+m+n;
	scalar_t multiL,multiR;
	if (n>=m){
		multiL=1;
		multiR=n/m;
	}else{
		multiL=m/n;
		multiR=1;
	}
	const int Block=1024;
	__shared__ scalar_t buf[Block*3];
	for (int i=blockIdx.x;i<b;i+=gridDim.x){
		for (int j=threadIdx.x;j<n*m;j+=blockDim.x)
			match[i*n*m+j]=0;
		for (int j=threadIdx.x;j<n;j+=blockDim.x)
			remainL[j]=multiL;
		for (int j=threadIdx.x;j<m;j+=blockDim.x)
			remainR[j]=multiR;
		__syncthreads();
		for (int j=7;j>=-2;j--){
			scalar_t level=-powf(4.0f,j);
			if (j==-2){
				level=0;
			}
			for (int k0=0;k0<n;k0+=blockDim.x){
				int k=k0+threadIdx.x;
				scalar_t x1=0,y1=0,z1=0;
				if (k<n){
					x1=p1[i*n*2+k*2+0];
					y1=p1[i*n*2+k*2+1];
				}
				scalar_t suml=1e-9f;
				for (int l0=0;l0<m;l0+=Block){
					int lend=min(m,l0+Block)-l0;
					for (int l=threadIdx.x;l<lend;l+=blockDim.x){
						scalar_t x2=p2[i*m*2+l0*2+l*2+0];
						scalar_t y2=p2[i*m*2+l0*2+l*2+1];
						buf[l*3+0]=x2;
						buf[l*3+1]=y2;
						buf[l*3+2]=remainR[l0+l];
					}
					__syncthreads();
					for (int l=0;l<lend;l++){
						scalar_t x2=buf[l*3+0];
						scalar_t y2=buf[l*3+1];
						scalar_t d=level*((x2-x1)*(x2-x1)+(y2-y1)*(y2-y1));
						scalar_t w=__expf(d)*buf[l*3+2];
						suml+=w;
					}
					__syncthreads();
				}
				if (k<n)
					ratioL[k]=remainL[k]/suml;
			}
			__syncthreads();
			for (int l0=0;l0<m;l0+=blockDim.x){
				int l=l0+threadIdx.x;
				scalar_t x2=0,y2=0;
				if (l<m){
					x2=p2[i*m*2+l*2+0];
					y2=p2[i*m*2+l*2+1];
				}
				scalar_t sumr=0;
				for (int k0=0;k0<n;k0+=Block){
					int kend=min(n,k0+Block)-k0;
					for (int k=threadIdx.x;k<kend;k+=blockDim.x){
						buf[k*3+0]=p1[i*n*2+k0*2+k*2+0];
						buf[k*3+1]=p1[i*n*2+k0*2+k*2+1];
						buf[k*3+2]=ratioL[k0+k];
					}
					__syncthreads();
					for (int k=0;k<kend;k++){
						scalar_t x1=buf[k*3+0];
						scalar_t y1=buf[k*3+1];
						scalar_t w=__expf(level*((x2-x1)*(x2-x1)+(y2-y1)*(y2-y1)))*buf[k*3+2];
						sumr+=w;
					}
					__syncthreads();
				}
				if (l<m){
					sumr*=remainR[l];
					scalar_t consumption=fminf(remainR[l]/(sumr+1e-9f),1.0f);
					ratioR[l]=consumption*remainR[l];
					remainR[l]=fmaxf(0.0f,remainR[l]-sumr);
				}
			}
			__syncthreads();
			for (int k0=0;k0<n;k0+=blockDim.x){
				int k=k0+threadIdx.x;
				scalar_t x1=0,y1=0,z1=0;
				if (k<n){
					x1=p1[i*n*2+k*2+0];
					y1=p1[i*n*2+k*2+1];
				}
				scalar_t suml=0;
				for (int l0=0;l0<m;l0+=Block){
					int lend=min(m,l0+Block)-l0;
					for (int l=threadIdx.x;l<lend;l+=blockDim.x){
						buf[l*3+0]=p2[i*m*2+l0*2+l*2+0];
						buf[l*3+1]=p2[i*m*2+l0*2+l*2+1];
						buf[l*3+2]=ratioR[l0+l];
					}
					__syncthreads();
					scalar_t rl=ratioL[k];
					if (k<n){
						for (int l=0;l<lend;l++){
							scalar_t x2=buf[l*3+0];
							scalar_t y2=buf[l*3+1];
							scalar_t w=__expf(level*((x2-x1)*(x2-x1)+(y2-y1)*(y2-y1)))*rl*buf[l*3+2];
							match[i*n*m+(l0+l)*n+k]+=w;
							suml+=w;
						}
					}
					__syncthreads();
				}
				if (k<n)
					remainL[k]=fmaxf(0.0f,remainL[k]-suml);
			}
			__syncthreads();
		}
	}
}

//void approxmatchLauncher(int b,int n,int m,const scalar_t * p1,const scalar_t * p2,scalar_t * match,scalar_t * temp){
//	approxmatch<<<32,512>>>(b,n,m,p1,p2,match,temp);
//}

/* ApproxMatch forward interface
Input:
  p1: (B, N1, 3)  # dataset_points
  p2: (B, N2, 3)  # query_points
Output:
  match: (B, N2, N1)
*/
at::Tensor ApproxMatchForward(
    const at::Tensor p1,
    const at::Tensor p2){
  const auto b = p1.size(0);
  const auto n = p1.size(1);
  const auto m = p2.size(1);

  CHECK_EQ(p2.size(0), b);
  CHECK_EQ(p1.size(2), 2);
  CHECK_EQ(p2.size(2), 2);
  CHECK_INPUT(p1);
  CHECK_INPUT(p2);

  auto match = at::zeros({b, m, n}, p1.type());
  auto temp = at::zeros({b, (n+m)*2}, p1.type());

  AT_DISPATCH_FLOATING_TYPES(p1.scalar_type(), "ApproxMatchForward", ([&] {
        approxmatch<scalar_t><<<32,512>>>(b, n, m, p1.data<scalar_t>(), p2.data<scalar_t>(), match.data<scalar_t>(), temp.data<scalar_t>());
  }));
  THCudaCheck(hipGetLastError());

  return match;
}


/********************************
* Forward kernel for matchcost
*********************************/

template<typename scalar_t>
__global__ void matchcost(int b,int n,int m,const scalar_t * __restrict__ p1,const scalar_t * __restrict__ p2,const scalar_t * __restrict__ match,scalar_t * __restrict__ out){
	__shared__ scalar_t allsum[512];
	const int Block=1024;
	__shared__ scalar_t buf[Block*2];
	for (int i=blockIdx.x;i<b;i+=gridDim.x){
		scalar_t subsum=0;
		for (int k0=0;k0<n;k0+=blockDim.x){
			int k=k0+threadIdx.x;
			scalar_t x1=0,y1=0;
			if (k<n){
				x1=p1[i*n*2+k*2+0];
				y1=p1[i*n*2+k*2+1];
			}
			for (int l0=0;l0<m;l0+=Block){
				int lend=min(m,l0+Block)-l0;
				for (int l=threadIdx.x;l<lend*2;l+=blockDim.x)
					buf[l]=p2[i*m*2+l0*2+l];
				__syncthreads();
				if (k<n){
					for (int l=0;l<lend;l++){
						scalar_t x2=buf[l*2+0];
						scalar_t y2=buf[l*2+1];
						scalar_t d=(x2-x1)*(x2-x1)+(y2-y1)*(y2-y1);
						subsum+=d*match[i*n*m+(l0+l)*n+k];
					}
				}
				__syncthreads();
			}
		}
		allsum[threadIdx.x]=subsum;
		for (int j=1;j<blockDim.x;j<<=1){
			__syncthreads();
			if ((threadIdx.x&j)==0 && threadIdx.x+j<blockDim.x){
				allsum[threadIdx.x]+=allsum[threadIdx.x+j];
			}
		}
		if (threadIdx.x==0)
			out[i]=allsum[0];
		__syncthreads();
	}
}

//void matchcostLauncher(int b,int n,int m,const scalar_t * p1,const scalar_t * p2,const scalar_t * match,scalar_t * out){
//	matchcost<<<32,512>>>(b,n,m,p1,p2,match,out);
//}

/* MatchCost forward interface
Input:
  p1: (B, N1, 3)  # dataset_points
  p2: (B, N2, 3)  # query_points
  match: (B, N2, N1)
Output:
  cost: (B)
*/
at::Tensor MatchCostForward(
    const at::Tensor p1,
    const at::Tensor p2,
    const at::Tensor match){
  const auto b = p1.size(0);
  const auto n = p1.size(1);
  const auto m = p2.size(1);

  CHECK_EQ(p2.size(0), b);
  CHECK_EQ(p1.size(2), 2);
  CHECK_EQ(p2.size(2), 2);
  CHECK_INPUT(p1);
  CHECK_INPUT(p2);

  auto cost = at::zeros({b}, p1.type());

  AT_DISPATCH_FLOATING_TYPES(p1.scalar_type(), "MatchCostForward", ([&] {
        matchcost<scalar_t><<<32,512>>>(b, n, m, p1.data<scalar_t>(), p2.data<scalar_t>(), match.data<scalar_t>(), cost.data<scalar_t>());
  }));
  THCudaCheck(hipGetLastError());

  return cost;
}


/********************************
* matchcostgrad2 kernel
*********************************/

template<typename scalar_t>
__global__ void matchcostgrad2(int b,int n,int m,const scalar_t * __restrict__ grad_cost,const scalar_t * __restrict__ p1,const scalar_t * __restrict__ p2,const scalar_t * __restrict__ match,scalar_t * __restrict__ grad2){
	__shared__ scalar_t sum_grad[256*2];
	for (int i=blockIdx.x;i<b;i+=gridDim.x){
		int kbeg=m*blockIdx.y/gridDim.y;
		int kend=m*(blockIdx.y+1)/gridDim.y;
		for (int k=kbeg;k<kend;k++){
			scalar_t x2=p2[(i*m+k)*2+0];
			scalar_t y2=p2[(i*m+k)*2+1];
			scalar_t subsumx=0,subsumy=0;
			for (int j=threadIdx.x;j<n;j+=blockDim.x){
				scalar_t x1=x2-p1[(i*n+j)*2+0];
				scalar_t y1=y2-p1[(i*n+j)*2+1];
				scalar_t d=match[i*n*m+k*n+j]*2;
				subsumx+=x1*d;
				subsumy+=y1*d;
			}
			sum_grad[threadIdx.x*2+0]=subsumx;
			sum_grad[threadIdx.x*2+1]=subsumy;
			for (int j=1;j<blockDim.x;j<<=1){
				__syncthreads();
				int j1=threadIdx.x;
				int j2=threadIdx.x+j;
				if ((j1&j)==0 && j2<blockDim.x){
					sum_grad[j1*2+0]+=sum_grad[j2*2+0];
					sum_grad[j1*2+1]+=sum_grad[j2*2+1];
				}
			}
			if (threadIdx.x==0){
				grad2[(i*m+k)*2+0]=sum_grad[0]*grad_cost[i];
				grad2[(i*m+k)*2+1]=sum_grad[1]*grad_cost[i];
			}
			__syncthreads();
		}
	}
}

/********************************
* matchcostgrad1 kernel
*********************************/

template<typename scalar_t>
__global__ void matchcostgrad1(int b,int n,int m,const scalar_t * __restrict__ grad_cost,const scalar_t * __restrict__ p1,const scalar_t * __restrict__ p2,const scalar_t * __restrict__ match,scalar_t * __restrict__ grad1){
	for (int i=blockIdx.x;i<b;i+=gridDim.x){
		for (int l=threadIdx.x;l<n;l+=blockDim.x){
			scalar_t x1=p1[i*n*2+l*2+0];
			scalar_t y1=p1[i*n*2+l*2+1];
			scalar_t dx=0,dy=0,dz=0;
			for (int k=0;k<m;k++){
				scalar_t x2=p2[i*m*2+k*2+0];
				scalar_t y2=p2[i*m*2+k*2+1];
				scalar_t d=match[i*n*m+k*n+l]*2;
				dx+=(x1-x2)*d;
				dy+=(y1-y2)*d;
			}
			grad1[i*n*2+l*2+0]=dx*grad_cost[i];
			grad1[i*n*2+l*2+1]=dy*grad_cost[i];
		}
	}
}

//void matchcostgradLauncher(int b,int n,int m,const scalar_t * p1,const scalar_t * p2,const scalar_t * match,scalar_t * grad1,scalar_t * grad2){
//	matchcostgrad1<<<32,512>>>(b,n,m,p1,p2,match,grad1);
//	matchcostgrad2<<<dim3(32,32),256>>>(b,n,m,p1,p2,match,grad2);
//}


/* MatchCost backward interface
Input:
  grad_cost: (B)    # gradients on cost
  p1: (B, N1, 3)  # dataset_points
  p2: (B, N2, 3)  # query_points
  match: (B, N2, N1)
Output:
  grad1: (B, N1, 3)
  grad2: (B, N2, 3)
*/
std::vector<at::Tensor> MatchCostBackward(
    const at::Tensor grad_cost,
    const at::Tensor p1,
    const at::Tensor p2,
    const at::Tensor match){
  const auto b = p1.size(0);
  const auto n = p1.size(1);
  const auto m = p2.size(1);

  CHECK_EQ(p2.size(0), b);
  CHECK_EQ(p1.size(2), 2);
  CHECK_EQ(p2.size(2), 2);
  CHECK_INPUT(p1);
  CHECK_INPUT(p2);

  auto grad1 = at::zeros({b, n, 2}, p1.type());
  auto grad2 = at::zeros({b, m, 2}, p1.type());

  AT_DISPATCH_FLOATING_TYPES(p1.scalar_type(), "MatchCostBackward", ([&] {
        matchcostgrad1<scalar_t><<<32,512>>>(b, n, m, grad_cost.data<scalar_t>(), p1.data<scalar_t>(), p2.data<scalar_t>(), match.data<scalar_t>(), grad1.data<scalar_t>());
        matchcostgrad2<scalar_t><<<dim3(32,32),256>>>(b, n, m, grad_cost.data<scalar_t>(), p1.data<scalar_t>(), p2.data<scalar_t>(), match.data<scalar_t>(), grad2.data<scalar_t>());
  }));
  THCudaCheck(hipGetLastError());

  return std::vector<at::Tensor>({grad1, grad2});
}

#endif
