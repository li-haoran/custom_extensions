#include "hip/hip_runtime.h"
/*!
 *****************
 * COPYRIGHT
 *
 * LICENSE
 *
 * author: haoran li

 */

// modify from
// https://github.com/chengdazhi/Deformable-Convolution-V2-PyTorch/blob/mmdetection/mmdet/ops/dcn/src/deform_conv_cuda_kernel.cu

#include <ATen/ATen.h>
#include <float.h>
#include <math.h>
#include <stdio.h>
#include <THC/THCAtomics.cuh>

using namespace at;

#define CUDA_KERNEL_LOOP(i, n)                                 \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
       i += blockDim.x * gridDim.x)

const int CUDA_NUM_THREADS = 1024;
const int kMaxGridNum = 65535;
inline int GET_BLOCKS(const int N) {
  return std::min(kMaxGridNum, (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS);
}

template <typename scalar_t>
__device__ scalar_t offsetAccumulate_im2col_bilinear(
    const scalar_t* bottom_data,
    const int data_width,
    const int height,
    const int width,
    scalar_t h,
    scalar_t w) {
  int h_low = floor(h);
  int w_low = floor(w);
  int h_high = h_low + 1;
  int w_high = w_low + 1;

  scalar_t lh = h - h_low;
  scalar_t lw = w - w_low;
  scalar_t hh = 1 - lh, hw = 1 - lw;

  scalar_t v1 = 0;
  if (h_low >= 0 && w_low >= 0)
    v1 = bottom_data[h_low * data_width + w_low];
  scalar_t v2 = 0;
  if (h_low >= 0 && w_high <= width - 1)
    v2 = bottom_data[h_low * data_width + w_high];
  scalar_t v3 = 0;
  if (h_high <= height - 1 && w_low >= 0)
    v3 = bottom_data[h_high * data_width + w_low];
  scalar_t v4 = 0;
  if (h_high <= height - 1 && w_high <= width - 1)
    v4 = bottom_data[h_high * data_width + w_high];

  scalar_t w1 = hh * hw, w2 = hh * lw, w3 = lh * hw, w4 = lh * lw;

  scalar_t val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);
  return val;
}

template <typename scalar_t>
__device__ scalar_t get_gradient_weight(
    scalar_t argmax_h,
    scalar_t argmax_w,
    const int h,
    const int w,
    const int height,
    const int width) {
  if (argmax_h <= -1 || argmax_h >= height || argmax_w <= -1 ||
      argmax_w >= width) {
    // empty
    return 0;
  }

  int argmax_h_low = floor(argmax_h);
  int argmax_w_low = floor(argmax_w);
  int argmax_h_high = argmax_h_low + 1;
  int argmax_w_high = argmax_w_low + 1;

  scalar_t weight = 0;
  if (h == argmax_h_low && w == argmax_w_low)
    weight = (h + 1 - argmax_h) * (w + 1 - argmax_w);
  if (h == argmax_h_low && w == argmax_w_high)
    weight = (h + 1 - argmax_h) * (argmax_w + 1 - w);
  if (h == argmax_h_high && w == argmax_w_low)
    weight = (argmax_h + 1 - h) * (w + 1 - argmax_w);
  if (h == argmax_h_high && w == argmax_w_high)
    weight = (argmax_h + 1 - h) * (argmax_w + 1 - w);
  return weight;
}

template <typename scalar_t>
__device__ scalar_t get_coordinate_weight(
    scalar_t argmax_h,
    scalar_t argmax_w,
    const int height,
    const int width,
    const scalar_t ratio_h,
    const scalar_t ratio_w,
    const scalar_t* u_data,
    const scalar_t* v_data,
    const scalar_t top_grad_h,
    const scalar_t top_grad_w,
    const int data_width,
    int bp_dir) {
  if (argmax_h <= -1 || argmax_h >= height || argmax_w <= -1 ||
      argmax_w >= width) {
    // empty
    return 0;
  }

  int argmax_h_low = floor(argmax_h);
  int argmax_w_low = floor(argmax_w);
  int argmax_h_high = argmax_h_low + 1;
  int argmax_w_high = argmax_w_low + 1;

  scalar_t weight = 0;

  if (bp_dir == 0) {
    if (argmax_h_low >= 0 && argmax_w_low >= 0)
      weight += -1 * (argmax_w_low + 1 - argmax_w) *
          (u_data[argmax_h_low * data_width + argmax_w_low] * top_grad_h +
           v_data[argmax_h_low * data_width + argmax_w_low] * ratio_h /
               ratio_w * top_grad_w);
    if (argmax_h_low >= 0 && argmax_w_high <= width - 1)
      weight += -1 * (argmax_w - argmax_w_low) *
          (u_data[argmax_h_low * data_width + argmax_w_high] * top_grad_h +
           v_data[argmax_h_low * data_width + argmax_w_high] * ratio_h /
               ratio_w * top_grad_w);
    if (argmax_h_high <= height - 1 && argmax_w_low >= 0)
      weight += (argmax_w_low + 1 - argmax_w) *
          (u_data[argmax_h_high * data_width + argmax_w_low] * top_grad_h +
           v_data[argmax_h_high * data_width + argmax_w_low] * ratio_h /
               ratio_w * top_grad_w);
    if (argmax_h_high <= height - 1 && argmax_w_high <= width - 1)
      weight += (argmax_w - argmax_w_low) *
          (u_data[argmax_h_high * data_width + argmax_w_high] * top_grad_h +
           v_data[argmax_h_high * data_width + argmax_w_high] * ratio_h /
               ratio_w * top_grad_w);
  } else if (bp_dir == 1) {
    if (argmax_h_low >= 0 && argmax_w_low >= 0)
      weight += -1 * (argmax_h_low + 1 - argmax_h) *
          (u_data[argmax_h_low * data_width + argmax_w_low] * ratio_w /
               ratio_h * top_grad_h +
           v_data[argmax_h_low * data_width + argmax_w_low] * top_grad_w);
    if (argmax_h_low >= 0 && argmax_w_high <= width - 1)
      weight += (argmax_h_low + 1 - argmax_h) *
          (u_data[argmax_h_low * data_width + argmax_w_high] * ratio_w /
               ratio_h * top_grad_h +
           v_data[argmax_h_low * data_width + argmax_w_high] * top_grad_w);
    if (argmax_h_high <= height - 1 && argmax_w_low >= 0)
      weight += -1 * (argmax_h - argmax_h_low) *
          (u_data[argmax_h_high * data_width + argmax_w_low] * ratio_w /
               ratio_h * top_grad_h +
           v_data[argmax_h_high * data_width + argmax_w_low] * top_grad_w);
    if (argmax_h_high <= height - 1 && argmax_w_high <= width - 1)
      weight += (argmax_h - argmax_h_low) *
          (u_data[argmax_h_high * data_width + argmax_w_high] * ratio_w /
               ratio_h * top_grad_h +
           v_data[argmax_h_high * data_width + argmax_w_high] * top_grad_w);
  }

  return weight;
}

template <typename scalar_t>
__global__ void offsetAccumulate_im2col_gpu_kernel(
    const int n,
    const scalar_t* data_dcn,
    const scalar_t* data_target,
    const int num_target,
    const int num_offset,
    const int height_in,
    const int width_in,
    const int height_out,
    const int width_out,
    const scalar_t ratio_h,
    const scalar_t ratio_w,
    scalar_t* data_col) {
  CUDA_KERNEL_LOOP(index, n) {
    // index index of output matrix
    const int w_col = index % width_out;
    const int h_col = (index / width_out) % height_out;
    const int c_points = index / width_out / height_out;
    const int c_col = c_points * 2;
    const int c_target = c_points / num_offset;
    const int c_dcn = c_points % num_offset;

    scalar_t* data_col_ptr =
        data_col + (c_col * height_out + h_col) * width_out + w_col;
    // const scalar_t* data_im_ptr = data_im + ((b_col * num_channels + c_im) *
    // height + h_in) * width + w_in;
    const scalar_t* data_dcn_ptr = data_dcn + c_dcn * 2 * height_in * width_in;
    const scalar_t* data_target_ptr =
        data_target + (c_target * 2 * height_out + h_col) * width_out + w_col;

    const int data_target_h_ptr = 0;
    const int data_target_w_ptr = height_out * width_out;
    const scalar_t offset_h = data_target_ptr[data_target_h_ptr];
    const scalar_t offset_w = data_target_ptr[data_target_w_ptr];
    scalar_t val_h = static_cast<scalar_t>(0);
    scalar_t val_w = static_cast<scalar_t>(0);

    const scalar_t h_im = (h_col + offset_h) * ratio_h;
    const scalar_t w_im = (w_col + offset_w) * ratio_w;
    if (h_im > -1 && w_im > -1 && h_im < height_in && w_im < width_in) {
      const scalar_t* data_dcn_ptr_h = data_dcn_ptr;
      val_h = offsetAccumulate_im2col_bilinear(
          data_dcn_ptr_h, width_in, height_in, width_in, h_im, w_im);
      const scalar_t* data_dcn_ptr_w = data_dcn_ptr + height_in * width_in;
      val_w = offsetAccumulate_im2col_bilinear(
          data_dcn_ptr_w, width_in, height_in, width_in, h_im, w_im);
    }
    scalar_t* data_col_ptr_h = data_col_ptr;
    *data_col_ptr_h = val_h / ratio_h + offset_h;
    scalar_t* data_col_ptr_w = data_col_ptr + height_out * width_out;
    *data_col_ptr_w = val_w / ratio_w + offset_w;
  }
}

void offsetAccumulate_im2col(
    const at::Tensor data_dcn,
    const at::Tensor data_target,
    const int num_target,
    const int num_offset,
    const int height_out,
    const int width_out,
    const int height_in,
    const int width_in,
    float ratio_h,
    float ratio_w,
    at::Tensor data_col) {
  // num_axes should be smaller than block size
  // todo: check parallel_imgs is correctly passed in
  int num_kernels = num_target * num_offset * height_out * width_out;

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      data_col.type(), "offsetAccumulate_im2col_gpu", ([&] {
        const scalar_t* data_dcn_ = data_dcn.data<scalar_t>();
        const scalar_t* data_target_ = data_target.data<scalar_t>();
        const scalar_t ratio_h_ = static_cast<scalar_t>(ratio_h);
        const scalar_t ratio_w_ = static_cast<scalar_t>(ratio_w);
        scalar_t* data_col_ = data_col.data<scalar_t>();

        offsetAccumulate_im2col_gpu_kernel<<<
            GET_BLOCKS(num_kernels),
            CUDA_NUM_THREADS>>>(
            num_kernels,
            data_dcn_,
            data_target_,
            num_target,
            num_offset,
            height_in,
            width_in,
            height_out,
            width_out,
            ratio_h_,
            ratio_w_,
            data_col_);
      }));

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in offsetAccumulate_im2col: %s\n", hipGetErrorString(err));
  }
}

template <typename scalar_t>
__global__ void offsetAccumulate_col2im_gpu_kernel(
    const int n,
    const scalar_t* grad_output,
    const scalar_t* data_target,
    const int num_target,
    const int num_offset,
    const int height_in,
    const int width_in,
    const int height_out,
    const int width_out,
    const scalar_t ratio_h,
    const scalar_t ratio_w,
    scalar_t* grad_dcn_offset) {
  CUDA_KERNEL_LOOP(index, n) {
    const int w_col = index % width_out;
    const int h_col = (index / width_out) % height_out;
    const int m_col = index / width_out / height_out % num_offset;
    const int n_col = index / width_out / height_out / num_offset;
    // compute the start and end of the output

    const int grad_output_index_base =
        (2 * m_col * height_out + h_col) * width_out + w_col;

    const int plane_num = height_out * width_out;
    const int step = 2 * n_col * plane_num;

    const scalar_t* data_grad_output_ptr = grad_output + grad_output_index_base;

    const scalar_t* data_target_ptr = data_target + h_col * width_out + w_col;

    scalar_t* data_grad_dcn_offset_ptr =
        grad_dcn_offset + 2 * m_col * height_in * width_in;

    const int index_h = step * num_offset;
    const int index_w = step * num_offset + plane_num;
    const scalar_t cur_top_grad_h = data_grad_output_ptr[index_h];
    const scalar_t cur_top_grad_w = data_grad_output_ptr[index_w];

    const scalar_t cur_target_h = data_target_ptr[step];
    const scalar_t cur_target_w = data_target_ptr[step + plane_num];

    const scalar_t cur_inv_h_data = (h_col + cur_target_h) * ratio_h;
    const scalar_t cur_inv_w_data = (w_col + cur_target_w) * ratio_w;

    const int cur_h = (int)cur_inv_h_data;
    const int cur_w = (int)cur_inv_w_data;

    for (int dy = -2; dy <= 2; dy++) {
      for (int dx = -2; dx <= 2; dx++) {
        if (cur_h + dy >= 0 && cur_h + dy < height_in && cur_w + dx >= 0 &&
            cur_w + dx < width_in && abs(cur_inv_h_data - (cur_h + dy)) < 1 &&
            abs(cur_inv_w_data - (cur_w + dx)) < 1) {
          int cur_bottom_grad_pos_h = (cur_h + dy) * width_in + cur_w + dx;
          scalar_t weight = get_gradient_weight(
              cur_inv_h_data,
              cur_inv_w_data,
              cur_h + dy,
              cur_w + dx,
              height_in,
              width_in);

          scalar_t this_grad_h = 1.0 / ratio_h * weight * cur_top_grad_h;
          atomicAdd(
              data_grad_dcn_offset_ptr + cur_bottom_grad_pos_h, this_grad_h);

          int cur_bottom_grad_pos_w =
              height_in * width_in + cur_bottom_grad_pos_h;
          scalar_t this_grad_w = 1.0 / ratio_w * weight * cur_top_grad_w;
          atomicAdd(
              data_grad_dcn_offset_ptr + cur_bottom_grad_pos_w, this_grad_w);
        }
      }
    }
  }
}

void offsetAccumulate_col2im(
    const at::Tensor grad_output,
    const at::Tensor data_target,
    const int num_target,
    const int num_offset,
    const int height_out,
    const int width_out,
    const int height_in,
    const int width_in,
    float ratio_h,
    float ratio_w,
    at::Tensor grad_dcn_offset) {
  int num_kernels = num_target * num_offset * height_out * width_out;

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      grad_output.type(), "offsetAccumulate_col2im_gpu", ([&] {
        const scalar_t* grad_output_ = grad_output.data<scalar_t>();
        const scalar_t* data_target_ = data_target.data<scalar_t>();
        scalar_t* grad_dcn_offset_ = grad_dcn_offset.data<scalar_t>();
        const scalar_t ratio_h_ = static_cast<scalar_t>(ratio_h);
        const scalar_t ratio_w_ = static_cast<scalar_t>(ratio_w);

        offsetAccumulate_col2im_gpu_kernel<<<
            GET_BLOCKS(num_kernels),
            CUDA_NUM_THREADS>>>(
            num_kernels,
            grad_output_,
            data_target_,
            num_target,
            num_offset,
            height_in,
            width_in,
            height_out,
            width_out,
            ratio_h_,
            ratio_w_,
            grad_dcn_offset_);
      }));

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf(
        "error in offsetAccumulate_col2im: %s\n",
        hipGetErrorString(err));
  }
}

template <typename scalar_t>
__global__ void offsetAccumulate_col2im_coord_gpu_kernel(
    const int n,
    const scalar_t* grad_output,
    const scalar_t* data_dcn,
    const scalar_t* data_target,
    const int num_target,
    const int num_offset,
    const int height_in,
    const int width_in,
    const int height_out,
    const int width_out,
    const scalar_t ratio_h,
    const scalar_t ratio_w,
    scalar_t* grad_target_offset) {
  CUDA_KERNEL_LOOP(index, n) {
    const int w_col = index % width_out;
    const int h_col = (index / width_out) % height_out;
    const int m_col = index / width_out / height_out % num_offset;
    const int n_col = index / width_out / height_out / num_offset;
    // compute the start and end of the output

    const int grad_output_index_base =
        (2 * m_col * height_out + h_col) * width_out + w_col;

    const int plane_num = height_out * width_out;
    const int step = 2 * n_col * plane_num;

    const scalar_t* data_grad_output_ptr = grad_output + grad_output_index_base;
    const scalar_t* data_target_ptr = data_target + h_col * width_out + w_col;
    const int index_h = step * num_offset;
    const int index_w = step * num_offset + plane_num;
    const scalar_t cur_top_grad_h = data_grad_output_ptr[index_h];
    const scalar_t cur_top_grad_w = data_grad_output_ptr[index_w];

    const scalar_t cur_target_h = data_target_ptr[step];
    const scalar_t cur_target_w = data_target_ptr[step + plane_num];

    scalar_t cur_inv_h_data = (h_col + cur_target_h) * ratio_h;
    scalar_t cur_inv_w_data = (w_col + cur_target_w) * ratio_w;

    const scalar_t* u_data_ptr = data_dcn + 2 * m_col * height_in * width_in;
    const scalar_t* v_data_ptr =
        data_dcn + (2 * m_col + 1) * height_in * width_in;

    scalar_t* data_grad_target_offset_ptr = grad_target_offset +
        2 * n_col * height_out * width_out + h_col * width_out + w_col;
    if (cur_inv_h_data <= -1 || cur_inv_w_data <= -1 ||
        cur_inv_h_data >= height_in || cur_inv_w_data >= width_in) {
      cur_inv_h_data = cur_inv_w_data = -2;
    }
    const scalar_t weight_h = get_coordinate_weight(
        cur_inv_h_data,
        cur_inv_w_data,
        height_in,
        width_in,
        ratio_h,
        ratio_w,
        u_data_ptr,
        v_data_ptr,
        cur_top_grad_h,
        cur_top_grad_w,
        width_in,
        0);

    scalar_t val_h = cur_top_grad_h + weight_h;
    atomicAdd(data_grad_target_offset_ptr, val_h);

    const scalar_t weight_w = get_coordinate_weight(
        cur_inv_h_data,
        cur_inv_w_data,
        height_in,
        width_in,
        ratio_h,
        ratio_w,
        u_data_ptr,
        v_data_ptr,
        cur_top_grad_h,
        cur_top_grad_w,
        width_in,
        1);

    scalar_t val_w = cur_top_grad_w + weight_w;
    atomicAdd(data_grad_target_offset_ptr + height_out * width_out, val_w);
  }
}

void offsetAccumulate_col2im_coord(
    const at::Tensor grad_output,
    const at::Tensor data_dcn,
    const at::Tensor data_target,
    const int num_target,
    const int num_offset,
    const int height_out,
    const int width_out,
    const int height_in,
    const int width_in,
    float ratio_h,
    float ratio_w,
    at::Tensor grad_target_offset) {
  int num_kernels = num_target * num_offset * height_out * width_out;

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      grad_output.type(), "offsetAccumulate_col2im_coord_gpu", ([&] {
        const scalar_t* grad_output_ = grad_output.data<scalar_t>();
        const scalar_t* data_target_ = data_target.data<scalar_t>();
        const scalar_t* data_dcn_ = data_dcn.data<scalar_t>();
        const scalar_t ratio_h_ = static_cast<scalar_t>(ratio_h);
        const scalar_t ratio_w_ = static_cast<scalar_t>(ratio_w);
        scalar_t* grad_target_offset_ = grad_target_offset.data<scalar_t>();

        offsetAccumulate_col2im_coord_gpu_kernel<<<
            GET_BLOCKS(num_kernels),
            CUDA_NUM_THREADS>>>(
            num_kernels,
            grad_output_,
            data_dcn_,
            data_target_,
            num_target,
            num_offset,
            height_in,
            width_in,
            height_out,
            width_out,
            ratio_h_,
            ratio_w_,
            grad_target_offset_);
      }));

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf(
        "error in offsetAccumulate_col2im_coord: %s\n",
        hipGetErrorString(err));
  }
}
